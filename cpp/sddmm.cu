
#include <hip/hip_runtime.h>
#include <chrono>
#include <hip/hip_runtime_api.h>  // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>          // cusparseSpMM
#include <iostream>
#include <random>
#include <stdio.h>   // printf
#include <stdlib.h>  // EXIT_FAILURE

#define CHECK_CUDA(func)                                         \
  {                                                              \
    hipError_t status = (func);                                 \
    if (status != hipSuccess) {                                 \
      printf("CUDA API failed at line %d with error: %s (%d)\n", \
             __LINE__,                                           \
             hipGetErrorString(status),                         \
             status);                                            \
      return;                                                    \
    }                                                            \
  }

#define CHECK_CUSPARSE(func)                                         \
  {                                                                  \
    hipsparseStatus_t status = (func);                                \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                         \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
             __LINE__,                                               \
             hipsparseGetErrorString(status),                         \
             status);                                                \
      return;                                                        \
    }                                                                \
  }
enum class TimeUnit {
  Second      = 0,
  MilliSecond = 3,
  MicroSecond = 6,
  NanoSecond  = 9,
};

template <typename Rep>
struct Timer {
  explicit Timer(TimeUnit tu = TimeUnit::MilliSecond) : tu_(tu) {}
  void start() { startRecord = std::chrono::steady_clock::now(); }
  void end() { endRecord = std::chrono::steady_clock::now(); }
  Rep getResult()
  {
    auto duration_ = std::chrono::duration_cast<std::chrono::nanoseconds>(endRecord - startRecord);
    auto pow_      = static_cast<int32_t>(tu_) - static_cast<int32_t>(TimeUnit::NanoSecond);
    auto factor    = static_cast<Rep>(std::pow(10, pow_));
    return static_cast<Rep>(duration_.count()) * factor;
  }

 private:
  TimeUnit tu_;
  std::chrono::time_point<std::chrono::steady_clock> startRecord{};
  std::chrono::time_point<std::chrono::steady_clock> endRecord{};
};

struct SDDMMBenchParams {
  size_t m;  // m parameter of the SDDMM
  size_t k;  // k parameter of the SDDMM
  size_t n;  // n parameter of the SDDMM
  float sparsity;
  float alpha = 1.0;
  float beta  = 0.0;
};

template <typename ValueType, typename IndexType = int64_t>
void convert_to_csr(std::vector<bool>& matrix,
                    size_t rows,
                    size_t cols,
                    std::vector<ValueType>& values,
                    std::vector<IndexType>& indices,
                    std::vector<IndexType>& indptr)
{
  indptr.push_back(0);

  for (size_t i = 0; i < rows; ++i) {
    for (size_t j = 0; j < cols; ++j) {
      if (matrix[i * cols + j]) {
        values.push_back(static_cast<ValueType>(1.0f));
        indices.push_back(static_cast<IndexType>(j));
      }
    }
    indptr.push_back(static_cast<IndexType>(values.size()));
  }
}

size_t create_sparse_matrix(size_t m, size_t n, float sparsity, std::vector<bool>& matrix)
{
  size_t total_elements = static_cast<size_t>(m * n);
  size_t num_ones       = static_cast<size_t>((total_elements * 1.0f) * sparsity);
  size_t res            = num_ones;

  for (size_t i = 0; i < total_elements; ++i) {
    matrix[i] = false;
  }

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<> dis(0, total_elements - 1);

  while (num_ones > 0) {
    size_t index = dis(gen);

    if (matrix[index] == false) {
      matrix[index] = true;
      num_ones--;
    }
  }
  return res;
}

void uniform(float* array, int size)
{
  std::random_device rd;
  std::mt19937 eng(123456ULL);
  std::uniform_real_distribution<> distr(-1.0f, 1.0f);

  std::vector<float> randomArray(size);

  for (int i = 0; i < size; ++i) {
    randomArray[i] = static_cast<float>(distr(eng));
  }
}

template <typename ValueType, typename IndexType = int64_t>
void convert_to_csr_problem(std::vector<bool>& matrix,
                    size_t rows,
                    size_t cols,
                    float* values,
                    IndexType* indices,
                    IndexType* indptr)
{
  IndexType offset_indptr = 0;
  IndexType offset_values = 0;
  indptr[offset_indptr++] = 0;

  for (size_t i = 0; i < rows; ++i) {
    for (size_t j = 0; j < cols; ++j) {
      if (matrix[i * cols + j]) {
        values[offset_values]  = static_cast<float>(1.0f);
        indices[offset_values] = static_cast<IndexType>(j);
        offset_values++;
      }
    }
    indptr[offset_indptr++] = static_cast<IndexType>(offset_values);
  }
}

void test_main(SDDMMBenchParams& params, Timer<double>& timer)
{
  // Host problem definition
  size_t lda    = params.k;
  size_t ldb    = params.k;
  size_t A_size = params.m * params.k;
  size_t B_size = params.k * params.n;
  size_t C_size = params.m * params.n;
  float* hA     = (float*)malloc(sizeof(float) * A_size);
  float* hB     = (float*)malloc(sizeof(float) * B_size);

  uniform(hA, A_size);
  uniform(hB, B_size);

  std::vector<bool> c_dense_data_h(C_size);

  size_t c_true_nnz = create_sparse_matrix(params.m, params.n, params.sparsity, c_dense_data_h);

  std::cout << "c_true_nnz: " << c_true_nnz << std::endl;

  std::vector<float> hC_values(c_true_nnz);
  std::vector<int64_t> hC_columns(c_true_nnz);
  std::vector<int64_t> hC_offsets(params.m + 1);

  convert_to_csr<float, int64_t>(
    c_dense_data_h, params.m, params.n, hC_values, hC_columns, hC_offsets);
  //--------------------------------------------------------------------------
  // Device memory management
  int64_t *dC_offsets, *dC_columns;
  float *dC_values, *dB, *dA;
  CHECK_CUDA(hipMalloc((void**)&dA, A_size * sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&dB, B_size * sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&dC_offsets, (params.m + 1) * sizeof(int64_t)));
  CHECK_CUDA(hipMalloc((void**)&dC_columns, c_true_nnz * sizeof(int64_t)));
  CHECK_CUDA(hipMalloc((void**)&dC_values, c_true_nnz * sizeof(float)));

  CHECK_CUDA(hipMemcpy(dA, hA, A_size * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dB, hB, B_size * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dC_offsets, hC_offsets.data(), (params.m + 1) * sizeof(int64_t), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dC_columns, hC_columns.data(), c_true_nnz * sizeof(int64_t), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dC_values, hC_values.data(), c_true_nnz * sizeof(float), hipMemcpyHostToDevice));
  //--------------------------------------------------------------------------
  // CUSPARSE APIs
  hipsparseHandle_t handle = NULL;
  hipsparseDnMatDescr_t matA, matB;
  hipsparseSpMatDescr_t matC;
  void* dBuffer     = NULL;
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseCreate(&handle))
  // Create dense matrix A
  CHECK_CUSPARSE(
    hipsparseCreateDnMat(&matA, params.m, params.k, lda, dA, HIP_R_32F, HIPSPARSE_ORDER_ROW))
  // Create dense matrix B
  CHECK_CUSPARSE(
    hipsparseCreateDnMat(&matB, params.k, params.n, ldb, dB, HIP_R_32F, HIPSPARSE_ORDER_COL))
  // Create sparse matrix C in CSR format
  CHECK_CUSPARSE(hipsparseCreateCsr(&matC,
                                   params.m,
                                   params.n,
                                   c_true_nnz,
                                   dC_offsets,
                                   dC_columns,
                                   dC_values,
                                   HIPSPARSE_INDEX_64I,
                                   HIPSPARSE_INDEX_64I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F))
  // execute SpMM
  hipStream_t stream;

  CHECK_CUDA(hipStreamCreate(&stream));
  CHECK_CUSPARSE(hipsparseSetStream(handle, stream));
  // allocate an external buffer if needed
  CHECK_CUSPARSE(hipsparseSDDMM_bufferSize(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &params.alpha,
                                          matA,
                                          matB,
                                          &params.beta,
                                          matC,
                                          HIP_R_32F,
                                          HIPSPARSE_SDDMM_ALG_DEFAULT,
                                          &bufferSize))
  CHECK_CUDA(hipStreamSynchronize(stream));
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize * 4))

  // execute preprocess (optional)
  //   CHECK_CUSPARSE(cusparseSDDMM_preprocess(handle,
  //                                           CUSPARSE_OPERATION_NON_TRANSPOSE,
  //                                           CUSPARSE_OPERATION_NON_TRANSPOSE,
  //                                           &params.alpha,
  //                                           matA,
  //                                           matB,
  //                                           &params.beta,
  //                                           matC,
  //                                           CUDA_R_32F,
  //                                           CUSPARSE_SDDMM_ALG_DEFAULT,
  //                                           dBuffer))

  timer.start();
  CHECK_CUSPARSE(hipsparseSDDMM(handle,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &params.alpha,
                               matA,
                               matB,
                               &params.beta,
                               matC,
                               HIP_R_32F,
                               HIPSPARSE_SDDMM_ALG_DEFAULT,
                               dBuffer))

  CHECK_CUDA(hipStreamSynchronize(stream))
  timer.end();
  CHECK_CUDA(hipStreamDestroy(stream));

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE(hipsparseDestroyDnMat(matA))
  CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
  CHECK_CUSPARSE(hipsparseDestroy(handle))

  //--------------------------------------------------------------------------
  // device memory deallocation
  CHECK_CUDA(hipFree(dBuffer))
  CHECK_CUDA(hipFree(dA))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(dC_offsets))
  CHECK_CUDA(hipFree(dC_columns))
  CHECK_CUDA(hipFree(dC_values))

  free(hA);
  free(hB);
}

int main(void)
{
//   std::vector<SDDMMBenchParams> cases{{1024 * 1024, 128, 1024, 0.01, 1.0f, 0.0f}};
//
//   auto timer             = Timer<double>();
//   int times              = 3;
//   double accumulated_dur = 0.0;
//   for (auto params : cases) {
//     test_main(params, timer);
//     for (int time = 0; time < times; time++) {
//       test_main(params, timer);
//       accumulated_dur += timer.getResult();
//     }
//     std::cout << accumulated_dur / static_cast<double>(1.0 * times) << std::endl;
//   }

  std::vector<bool> c_dense_data_h { true, true, true,
                                     false, true, false,
                                     true, true, true,
                                     true, false, true};

  size_t c_true_nnz = 9;

  std::cout << "c_true_nnz: " << c_true_nnz << std::endl;

  std::vector<float> hC_values(c_true_nnz);
  std::vector<int64_t> hC_columns(c_true_nnz);
  std::vector<int64_t> hC_offsets(4 + 1);

  convert_to_csr_problem<float, int64_t>(
    c_dense_data_h, 4, 3, hC_values.data(), hC_columns.data(), hC_offsets.data());
  for(auto a: hC_values) std::cout << a << ", ";
  std::cout << std::endl;
  for(auto a: hC_columns) std::cout << a << ", ";
  std::cout << std::endl;
  for(auto a: hC_offsets) std::cout << a << ", ";
  std::cout << std::endl;

  return EXIT_SUCCESS;
}