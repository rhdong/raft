#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime_api.h>  // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>          // hipsparseSpMM
#include <random>
#include <stdio.h>   // printf
#include <stdlib.h>  // EXIT_FAILURE

#define CHECK_CUDA(func)                                         \
  {                                                              \
    hipError_t status = (func);                                 \
    if (status != hipSuccess) {                                 \
      printf("CUDA API failed at line %d with error: %s (%d)\n", \
             __LINE__,                                           \
             hipGetErrorString(status),                         \
             status);                                            \
      return EXIT_FAILURE;                                       \
    }                                                            \
  }

#define CHECK_CUSPARSE(func)                                         \
  {                                                                  \
    hipsparseStatus_t status = (func);                                \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                         \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
             __LINE__,                                               \
             hipsparseGetErrorString(status),                         \
             status);                                                \
      return EXIT_FAILURE;                                           \
    }                                                                \
  }

template <typename Rep>
struct Timer {
  explicit Timer(TimeUnit tu = TimeUnit::Second) : tu_(tu) {}
  void start() { startRecord = std::chrono::steady_clock::now(); }
  void end() { endRecord = std::chrono::steady_clock::now(); }
  Rep getResult()
  {
    auto duration_ = std::chrono::duration_cast<std::chrono::nanoseconds>(endRecord - startRecord);
    auto pow_      = static_cast<int32_t>(tu_) - static_cast<int32_t>(TimeUnit::NanoSecond);
    auto factor    = static_cast<Rep>(std::pow(10, pow_));
    return static_cast<Rep>(duration_.count()) * factor;
  }

 private:
  TimeUnit tu_;
  std::chrono::time_point<std::chrono::steady_clock> startRecord{};
  std::chrono::time_point<std::chrono::steady_clock> endRecord{};
};

struct SDDMMBenchParams {
  size_t m;  // m parameter of the SDDMM
  size_t k;  // k parameter of the SDDMM
  size_t n;  // n parameter of the SDDMM
  float sparsity;
  ValueType alpha = 1.0;
  ValueType beta  = 0.0;
};

void convert_to_csr(std::vector<bool>& matrix,
                    size_t rows,
                    size_t cols,
                    std::vector<ValueType>& values,
                    std::vector<IndexType>& indices,
                    std::vector<IndexType>& indptr)
{
  indptr.push_back(0);

  for (size_t i = 0; i < rows; ++i) {
    for (size_t j = 0; j < cols; ++j) {
      if (matrix[i * cols + j]) {
        values.push_back(static_cast<ValueType>(1.0f));
        indices.push_back(static_cast<IndexType>(j));
      }
    }
    indptr.push_back(static_cast<IndexType>(values.size()));
  }
}

size_t create_sparse_matrix(size_t m, size_t n, float sparsity, bool* matrix)
{
  size_t total_elements = static_cast<size_t>(m * n);
  size_t num_ones       = static_cast<size_t>((total_elements * 1.0f) * sparsity);
  size_t res            = num_ones;

  for (size_t i = 0; i < total_elements; ++i) {
    matrix[i] = false;
  }

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<> dis(0, total_elements - 1);

  while (num_ones > 0) {
    size_t index = dis(gen);

    if (matrix[index] == false) {
      matrix[index] = true;
      num_ones--;
    }
  }
  return res;
}

void uniform(float* array, int size)
{
  std::random_device rd;
  std::mt19937 eng(rd());
  std::uniform_real_distribution<> distr(0.0, 1.0);

  std::vector<float> randomArray(arraySize);

  for (int i = 0; i < size; ++i) {
    randomArray[i] = static_cast<float>(distr(eng));
  }
}

void test_main(SDDMMBenchParams& params, Timer<double>& timer)
{
  // Host problem definition
  int A_num_rows = params.m;
  int A_num_cols = params.k;
  int B_num_rows = A_num_cols;
  int B_num_cols = params.n;
  int lda        = A_num_cols;
  int ldb        = B_num_cols;
  int A_size     = lda * A_num_rows;
  int B_size     = ldb * B_num_rows;
  int C_size     = A_num_rows * B_num_cols;
  float hA*      = malloc(sizeof(float) * A_size);
  float hB*      = malloc(sizeof(float) * B_size);

  uniform(hA, A_size);
  uniform(hB, B_size);

  float c_dense_data_h* = malloc(sizeof(bool) * C_size);

  size_t c_true_nnz = create_sparse_matrix(A_num_rows, B_num_cols, params.sparsity, c_dense_data_h);

  int* hC_offsets  = malloc(sizeof(int) * (params.m + 1));
  int* hC_columns  = malloc(sizeof(int) * c_true_nnz);
  float* hC_values = malloc(sizeof(float) * c_true_nnz);

  convert_to_csr(c_dense_data_h, params.m, params.n, hC_values, hC_columns, hC_offsets);
  //--------------------------------------------------------------------------
  // Device memory management
  int *dC_offsets, *dC_columns;
  float *dC_values, *dB, *dA;
  CHECK_CUDA(hipMalloc((void**)&dA, A_size * sizeof(float)))
  CHECK_CUDA(hipMalloc((void**)&dB, B_size * sizeof(float)))
  CHECK_CUDA(hipMalloc((void**)&dC_offsets, (A_num_rows + 1) * sizeof(int)))
  CHECK_CUDA(hipMalloc((void**)&dC_columns, c_true_nnz * sizeof(int)))
  CHECK_CUDA(hipMalloc((void**)&dC_values, c_true_nnz * sizeof(float)))

  CHECK_CUDA(hipMemcpy(dA, hA, A_size * sizeof(float), hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dB, hB, B_size * sizeof(float), hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dC_offsets, hC_offsets, (A_num_rows + 1) * sizeof(int), hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dC_columns, hC_columns, c_true_nnz * sizeof(int), hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dC_values, hC_values, c_true_nnz * sizeof(float), hipMemcpyHostToDevice))
  //--------------------------------------------------------------------------
  // CUSPARSE APIs
  hipsparseHandle_t handle = NULL;
  hipsparseDnMatDescr_t matA, matB;
  hipsparseSpMatDescr_t matC;
  void* dBuffer     = NULL;
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseCreate(&handle))
  // Create dense matrix A
  CHECK_CUSPARSE(
    hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA, HIP_R_32F, HIPSPARSE_ORDER_ROW))
  // Create dense matrix B
  CHECK_CUSPARSE(
    hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB, HIP_R_32F, HIPSPARSE_ORDER_COL))
  // Create sparse matrix C in CSR format
  CHECK_CUSPARSE(hipsparseCreateCsr(&matC,
                                   A_num_rows,
                                   B_num_cols,
                                   c_true_nnz,
                                   dC_offsets,
                                   dC_columns,
                                   dC_values,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F))
  // allocate an external buffer if needed
  CHECK_CUSPARSE(hipsparseSDDMM_bufferSize(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &params.alpha,
                                          matA,
                                          matB,
                                          &params.beta,
                                          matC,
                                          HIP_R_32F,
                                          HIPSPARSE_SDDMM_ALG_DEFAULT,
                                          &bufferSize))
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

  // execute preprocess (optional)
  //   CHECK_CUSPARSE(hipsparseSDDMM_preprocess(handle,
  //                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
  //                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
  //                                           &alpha,
  //                                           matA,
  //                                           matB,
  //                                           &beta,
  //                                           matC,
  //                                           HIP_R_32F,
  //                                           HIPSPARSE_SDDMM_ALG_DEFAULT,
  //                                           dBuffer))
  // execute SpMM
  hipStream_t stream;

  CHECK_CUDA(hipStreamCreate(stream));
  timer.start();
  CHECK_CUSPARSE(hipsparseSDDMM(handle,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha,
                               matA,
                               matB,
                               &beta,
                               matC,
                               HIP_R_32F,
                               HIPSPARSE_SDDMM_ALG_DEFAULT,
                               dBuffer,
                               stream))

  CHECK_CUDA(hipStreamSynchronize(stream))
  timer.end();
  CHECK_CUDA(hipStreamDestroy(stream));

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE(hipsparseDestroyDnMat(matA))
  CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
  CHECK_CUSPARSE(hipsparseDestroy(handle))

  //--------------------------------------------------------------------------
  // device memory deallocation
  CHECK_CUDA(hipFree(dBuffer))
  CHECK_CUDA(hipFree(dA))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(dC_offsets))
  CHECK_CUDA(hipFree(dC_columns))
  CHECK_CUDA(hipFree(dC_values))

  free(hA);
  free(hB);
  free(c_dense_data_h);
  free(hC_offsets);
  free(hC_columns);
  free(hC_values);
}

int main(void)
{
  std::vector<SDDMMBenchParams> cases{{1024 * 1024, 128, 1024, 0.249, 1.0f, 0.0f},
                                      {1024 * 1024, 128, 1024, 0.251, 1.0f, 0.0f}};

  auto timer = Timer<double>();
  for (auto params : cases) {
    test_main(params, timer);
  }
  return EXIT_SUCCESS;
}